#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#define N 1024
#define MIN(x, y) (x) < (y) ? (x) : (y)
__device__ double atomicMin(double *address, double val) {
    unsigned long long int *address_as_ull =
        (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}
double CPU_reduction(double *arr, int n) {
    double ret = arr[0];
    for (int i = 1; i < n; i++) {
        ret = min(ret, arr[i]);
    }
    return ret;
}
void generate_random_doubles(double *arr, int n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<unsigned long long int> dist;
    for (int i = 0; i < n; ++i) {
        auto c = dist(gen);
        memcpy(&arr[i], &c, sizeof(double));
    }
}
__global__ void cuda_reduction(double *arr, int n, double *ret) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ double temp[N];
    if(id < N)
        temp[threadIdx.x] = arr[id];
    __syncthreads();
    atomicMin(&temp[0], temp[threadIdx.x]);
    __syncthreads();
    if(threadIdx.x == 0)
        ret[0] = temp[0];
}
int main() {
    double *ret = new double;
    double *arr = new double[N];

    double *arrDevice, *retDevice;

    generate_random_doubles(arr, N);
    std::cout << "Generated numbers:";
    for (int i = 0; i < N; i++) {
        std::cout << ' ' << arr[i];
    }
    std::cout << '\n';

    // hipMalloc and hipMemcpy is required
    hipMalloc(&arrDevice, sizeof(double) * N);
    hipMalloc(&retDevice, sizeof(double));
    hipMemcpy(arrDevice, arr, sizeof(double) * N, hipMemcpyHostToDevice);
    cuda_reduction<<<1, N>>>(arrDevice, N, retDevice);
    hipDeviceSynchronize();
    hipMemcpy(ret, retDevice, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "The minimum value: " << *ret << '\n';
 
    delete ret;
    delete[] arr;
    hipDeviceReset();
    return 0;
}