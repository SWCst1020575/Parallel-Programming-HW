
#include <hip/hip_runtime.h>
#include <omp.h>
#include <png.h>
#include <stdio.h>
#include <zlib.h>

#include <atomic>
#include <cassert>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iostream>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8

#define BLOCK 4096
#define THREAD 256
#define GRID 16
std::atomic_int passedH;
// clang-format off
__constant__ int mask[MASK_N][MASK_X][MASK_Y] = {
    {{ -1, -4, -6, -4, -1},
     { -2, -8,-12, -8, -2},
     {  0,  0,  0,  0,  0},
     {  2,  8, 12,  8,  2},
     {  1,  4,  6,  4,  1}},
    {{ -1, -2,  0,  2,  1},
     { -4, -8,  0,  8,  4},
     { -6,-12,  0, 12,  6},
     { -4, -8,  0,  8,  4},
     { -1, -2,  0,  2,  1}}
};
// clang-format on

int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width,
             unsigned* channels) {
    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8)) return 1; /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr) return 4; /* out of memory */

    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4; /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32 i, rowbytes;
    png_bytep row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int)png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char*)malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }
#pragma omp simd
    for (i = 0; i < *height; ++i) {
        row_pointers[i] = *image + i * rowbytes;
    }

    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width,
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 0);

    png_bytep row_ptr[height];
#pragma omp simd
    for (int i = 0; i < height; ++i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}
__global__ void sobel(unsigned char* s, unsigned char* t, unsigned height, unsigned width, unsigned channels) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // printf("threadId %d %d %d\nblockId %d %d %d\nblockDim %d %d %d\ngridDim %d %d %d\nID %d\n\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, idx);
    // printf("%d\n", idx);
    //__shared__ unsigned char sharedImg[7 * 2 * THREAD * 3];
    int j = 0;

    j = 0;
    // for (int i = 0; i < 3; i++) {
    /*sharedImg[channels * (width * j + threadIdx.x + i) + 2] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x - (3 - i)) + 2];
    sharedImg[channels * (width * j + threadIdx.x + i) + 1] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x - (3 - i)) + 1];
    sharedImg[channels * (width * j + threadIdx.x + i)] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x - (3 - i))];
    sharedImg[channels * (width * j + threadIdx.x + 3 + THREAD + i) + 2] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x + 3 + THREAD + i) + 2];
    sharedImg[channels * (width * j + threadIdx.x + 3 + THREAD + i) + 1] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x + 3 + THREAD + i) + 1];
    sharedImg[channels * (width * j + threadIdx.x + 3 + THREAD + i)] = s[channels * (width * ((idx / width - 3 < 0) ? 0 : (idx / width - 3)) + threadIdx.x + 3 + THREAD + i)];
    j++;*/
    //}
    /*if (threadIdx.x == 1) {
        for (int i = 0; i < 7 * (THREAD + 6) * 3; i++)
            printf("%d ", sharedImg[i]);
        printf("\n");
    }*/

    //__syncthreads();
    int x, y, i, v, u;
    int R, G, B;
    float val[MASK_N * 3] = {0.0};
    int adjustX, adjustY, xBound, yBound;

    // float maxColor = 255.0;
    //  int x = threadIdx.x + blockIdx.x * blockDim.x;
    //   int y = threadIdx.y + blockIdx.y * blockDim.y;
    //  printf("%d %d\n",cudax,cuday);
    adjustX = (MASK_X % 2) ? 1 : 0;
    adjustY = (MASK_Y % 2) ? 1 : 0;
    xBound = MASK_X / 2;
    yBound = MASK_Y / 2;
    for (j = idx; j < height * width; j += BLOCK * THREAD) {
        /*int startH = ((idx / width - 3 < 0) ? 0 : (idx / width - 3));
        for (int jj = startH; jj < ((idx / width + 3 < height) ? (idx / width - 3) : height); jj++) {
            for (int c = 0; c < 2; c++) {
                if (channels * (width * jj + j + c - 3) / 2 >= 0) {
                    sharedImg[channels * (width * (jj - startH) + threadIdx.x + c) + 2] = s[channels * (width * jj + j + c - 3) + 2];
                    sharedImg[channels * (width * (jj - startH) + threadIdx.x + c) + 1] = s[channels * (width * jj + j + c - 3) + 1];
                    sharedImg[channels * (width * (jj - startH) + threadIdx.x + c)] = s[channels * (width * jj + j + c - 3) / 2];
                }
            }
            printf("C\n");
            // sharedImg[channels * (width * j + threadIdx.x + 3) + 1] = s[channels * (width * i + threadIdx.x) + 1];
            // sharedImg[channels * (width * j + threadIdx.x + 3)] = s[channels * (width * i + threadIdx.x)];
        }
        __syncthreads();*/
        x = j % width;
        y = j / width;
        for (i = 0; i < MASK_N; ++i) {
            val[i * 3 + 2] = 0.0;
            val[i * 3 + 1] = 0.0;
            val[i * 3] = 0.0;
            for (v = -yBound; v < yBound + adjustY; ++v) {
                for (u = -xBound; u < xBound + adjustX; ++u) {
                    if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                        R = s[channels * (width * (y + v) + (x + u)) + 2];
                        G = s[channels * (width * (y + v) + (x + u)) + 1];
                        B = s[channels * (width * (y + v) + (x + u)) + 0];
                        // R = sharedImg[channels * ((2 * THREAD) * (3 + v) + (threadIdx.x + u + 3)) + 2];
                        // G = sharedImg[channels * ((2 * THREAD) * (3 + v) + (threadIdx.x + u + 3)) + 1];
                        // B = sharedImg[channels * ((2 * THREAD) * (3 + v) + (threadIdx.x + u + 3))];
                        val[i * 3 + 2] += R * mask[i][u + xBound][v + yBound];
                        val[i * 3 + 1] += G * mask[i][u + xBound][v + yBound];
                        val[i * 3 + 0] += B * mask[i][u + xBound][v + yBound];
                    }
                }
            }
        }

        float totalR = 0.0;
        float totalG = 0.0;
        float totalB = 0.0;
        for (i = 0; i < MASK_N; ++i) {
            totalR += val[i * 3 + 2] * val[i * 3 + 2];
            totalG += val[i * 3 + 1] * val[i * 3 + 1];
            totalB += val[i * 3 + 0] * val[i * 3 + 0];
        }
        t[channels * (width * y + x) + 2] = (totalR > 4161600.0) ? 255 : sqrt(totalR) / SCALE;
        t[channels * (width * y + x) + 1] = (totalG > 4161600.0) ? 255 : sqrt(totalG) / SCALE;
        t[channels * (width * y + x) + 0] = (totalB > 4161600.0) ? 255 : sqrt(totalB) / SCALE;
    }
}

int main(int argc, char** argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char* src_img = NULL;
    passedH = 0;
    auto start = std::chrono::steady_clock::now();
    read_png(argv[1], &src_img, &height, &width, &channels);
    auto end = std::chrono::steady_clock::now();
    assert(channels == 3);
    const unsigned imgSize = height * width * channels * sizeof(unsigned char);
    unsigned char* dst_img = (unsigned char*)malloc(imgSize);
    unsigned char* src_imgCuda;
    unsigned char* dst_imgCuda;
    auto IOTime = end - start;
    start = std::chrono::steady_clock::now();
    hipMalloc(&src_imgCuda, imgSize);
    hipMemcpy(src_imgCuda, src_img, imgSize, hipMemcpyHostToDevice);
    hipMalloc(&dst_imgCuda, imgSize);
    // cudaMemcpy(dst_imgCuda, dst_img, imgSize, cudaMemcpyHostToDevice);
    end = std::chrono::steady_clock::now();
    auto MemTime = end - start;
    
    start = std::chrono::steady_clock::now();
    // dim3 threads(32, 16, 1);
    // dim3 blocks(width / 32 + 1, height / 16 + 1, 1);
    sobel<<<BLOCK, THREAD>>>(src_imgCuda, dst_imgCuda, height, width, channels);

    // sobel<<<1, THREAD>>>(src_imgCuda, dst_imgCuda, height, width, channels, i);
    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    auto KernelTime = end - start;
    start = std::chrono::steady_clock::now();
    hipMemcpy(dst_img, dst_imgCuda, imgSize, hipMemcpyDeviceToHost);
    end = std::chrono::steady_clock::now();
    MemTime += (end - start);
    start = std::chrono::steady_clock::now();
    write_png(argv[2], dst_img, height, width, channels);
    end = std::chrono::steady_clock::now();
    IOTime += (end - start);
    // free memory
    hipDeviceReset();
    free(src_img);
    free(dst_img);
    std::cout << "IO time: " << std::chrono::duration_cast<std::chrono::microseconds>(IOTime).count() << '\n';
    std::cout << "Memory time: " << std::chrono::duration_cast<std::chrono::microseconds>(MemTime).count() << '\n';
    std::cout << "Kernel time: " << std::chrono::duration_cast<std::chrono::microseconds>(KernelTime).count() << '\n';
    return 0;
}
